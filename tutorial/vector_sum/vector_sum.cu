#include "hip/hip_runtime.h"


#include <cassert>
#include <cstdlib>
#include <cstring>
#include <iostream>

// The size of the vector
#define NUM_DATA 1024

// Simple vector sum gpu kernel
__global__ void VecAddOnDevice(int *_c, int *_a, int *_b) {
  int tID = threadIdx.x;
  _c[tID] = _a[tID] + _b[tID];
}

/**
 * @brief Initialize host memory
 *
 * @param memory
 * @param size
 */
void HostMemInit(int *&memory, int size) {
  assert(size > 0);

  if (memory) {
    std::cout << "free" << std::endl;

    delete[] memory;
    memory = nullptr;
  }

  int memSize = sizeof(int) * size;
  // std::cout << NUM_DATA << " elements, " << memSize << " Bytes" << std::endl;

  memory = new int[size];
  memset(memory, 0, memSize);
}

/**
 * @brief Initialize device memory
 *
 * @param memory Pointer to the device memory to initialize
 * @param size Number of elements in the memory
 */
void DeviceMemInit(int *&memory, int size) {
  int memSize = sizeof(int) * size;
  hipError_t err = hipMalloc(&memory, memSize);
  if (err != hipSuccess) {
    std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
  hipMemset(memory, 0, memSize);
}

/**
 * @brief Return a random digit from 0 to 9
 *
 * @return int
 */
int RandDataGen() { return rand() % 10; }

/**
 * @brief Generate random data and fill the memory
 *
 * @param memory
 * @param size
 */
void DataPreFill(int *&memory, int size) {
  for (int i = 0; i < size; i++)
    memory[i] = RandDataGen();
}

/**
 * @brief Add two vectors and store the output to results on the host.
 *
 * @param result
 * @param vec_a
 * @param vec_b
 * @param size
 */
void VecAddOnHost(int *result, const int *vec_a, const int *vec_b, int size) {
  for (int i = 0; i < size; i++)
    result[i] = vec_a[i] + vec_b[i];
}

/**
 * @brief Check if two vectors are equal
 *
 * @param vec Pointer to the first vector
 * @param other Pointer to the second vector
 * @param size Number of elements in the vectors
 * @return true if the vectors are equal, false otherwise
 */
bool CheckVectorEqual(int *vec, int *other, int size) {
  for (int i = 0; i < size; i++) {
    if (vec[i] != other[i]) {
      std::cout << vec[i] << " != " << other[i] << std::endl;
      return false;
    }
  }
  return true;
}

/**
 * @brief Entry function
 *
 * @return int
 */
int main() {
  int *a = nullptr, *b = nullptr, *c = nullptr, *h_c = nullptr;
  int *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

  // Host-side computation
  // Memory allocation on the host-side
  HostMemInit(a, NUM_DATA);
  HostMemInit(b, NUM_DATA);
  HostMemInit(h_c, NUM_DATA);

  // Data gen and fill
  DataPreFill(a, NUM_DATA);
  DataPreFill(b, NUM_DATA);

  VecAddOnHost(h_c, a, b, NUM_DATA);

  // Device-side computation
  // Memory allocation to the device-side
  HostMemInit(c, NUM_DATA);
  DeviceMemInit(d_a, NUM_DATA);
  DeviceMemInit(d_b, NUM_DATA);
  DeviceMemInit(d_c, NUM_DATA);

  // Data Copy
  hipMemcpy(d_a, a, sizeof(int) * NUM_DATA, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(int) * NUM_DATA, hipMemcpyHostToDevice);

  // Kernel call
  VecAddOnDevice<<<1, NUM_DATA>>>(d_c, d_a, d_b);

  // Data copy-back
  hipMemcpy(c, d_c, sizeof(int) * NUM_DATA, hipMemcpyDeviceToHost);

  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Check the integrity of an output.
  bool res = CheckVectorEqual(c, h_c, NUM_DATA);

  if (res)
    std::cout << "GPU works well!" << std::endl;
  else
    std::cout << "GPU does not work well..." << std::endl;

  // Release host memory
  delete[] a;
  delete[] b;
  delete[] c;

  return 0;
}