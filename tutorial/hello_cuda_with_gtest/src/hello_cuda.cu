#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "hello_cuda.h"

__global__ void helloCUDA(void)
{
    printf("Hello CUDA from GPU!\n");
}

void launchKernel()
{
    helloCUDA<<<1, 10>>>();
    hipDeviceSynchronize();
}
