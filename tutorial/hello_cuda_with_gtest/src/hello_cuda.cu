#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hello_cuda.h"
#include <stdio.h>

__global__ void helloCUDA(void) { printf("Hello CUDA from GPU!\n"); }

void launchKernel() {
  helloCUDA<<<1, 10>>>();
  hipDeviceSynchronize();
}
